
#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;
int main(int argc, char const *argv[])
{

    int i,count;      
    hipDeviceProp_t prop;  
    hipError_t cudaStatus=hipGetDeviceCount(&count);  
    if(cudaStatus == hipSuccess) 
    {  
        cout<<"共有设备数目："<<count<<"\n";  
        if(count>0)  
        {  
            for(i=0;i<=count;i++)
            {  
                hipGetDeviceProperties(&prop,i);//获取设备的属性信息  
                cout<<"\n第"<<i+1<<"个设备信息：\n";  
                cout<<"设备名称："<<prop.name<<"\n";  
                cout<<"总内存："<<prop.totalGlobalMem/1048576<<"M\n";  
                cout<<"常量内存："<<prop.totalConstMem<<"字节\n";  
                cout<<"设备中处理器数目："<<prop.multiProcessorCount<<"个\n";  
                cout<<"每个线程块最多包含线程数目："<<prop.maxThreadsPerBlock<<"个\n";     
                cout<<"一个线程格中可包含的线程块数目：I="<< prop.maxGridSize[0]  
                    <<" J="<<prop.maxGridSize[1]<<" K="<<prop.maxGridSize[2]<<"\n";  
                cout<<"多维线程块中可以包含的最大线程数目：I="<< prop.maxThreadsDim[0]  
                    <<" J="<<prop.maxThreadsDim[1]<<" K="<<prop.maxThreadsDim[2]<<"\n";  
            }          
        }  
    else  
    {  
        cout<<"没有获取到设备信息！请检查计算机是否具有支持CUDA的显卡设备以及CUDA驱动程序版本是否需要更新！\n";  
    }     
}return 0;
}
