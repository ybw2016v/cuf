
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
struct soc
{
    float *va;
    int xplace;
    int yplace;
};


__device__ float * p0=NULL;
__device__ float * p1=NULL;
__device__ float * vx1=NULL;
__device__ float * vz1=NULL;
__device__ float * z0=NULL;

 float * P1=NULL;
float * VX1=NULL;
float * VZ1=NULL;
__device__ int xin=0;
__device__ int yin=0;
int xma=0;
int yma=0;
__device__ int size=0;
__device__ int m=8;
int SIZE=0;
int XIN=0;
int YIN=0;




extern "C" 
{
    int cucaldog(float * date,int xar,int yar,int xm,int ym);
    int sdogkel(int n);
    int sdoginit(float *pP1, float * pVX1,float * pVZ1,float *Z0,int xar,int yar,int xm,int ym,int n);
    // int soinit(float *f);
}

__global__ void zeroinit(float * date,int xr,int yr);
__global__ void jisuan1(float * p1j,float *vx1j,float* vz1j,float * z0j,int xr,int yr);
__global__ void jisuan2(float * p1j,float *vx1j,float* vz1j,float * z0j,int xr,int yr);
__global__ void jisuan3();
__global__ void sou(int x,int y,int n,float * p1j,float *vx1j,float* vz1j,float * z0j);
__global__ void zeroinit(float * date,int xr,int yr);
__global__ void kkp();
int sdoginit(float *pP1, float * pVX1,float * pVZ1,float *Z0,int xar,int yar,int xm,int ym,int n)
{
    xar=xar/sizeof(float);
    yar=yar/sizeof(float);
    xma=xm;
    
    yma=ym;
    
    XIN=xar;
    YIN=yar;
    // printf("%d %d\n",xin,yar);
    // cudaMemcpy(&xin,&XIN,sizeof(int),cudaMemcpyHostToDevice);
    memcpy(&xin,&XIN,sizeof(int));
    // cudaMemcpy(&yin,&YIN,sizeof(int),cudaMemcpyHostToDevice);
    memcpy(&yin,&YIN,sizeof(int));
    // printf("%d %d\n",xin,yar);
    SIZE=(ym*yar)*sizeof(float);
    memcpy(&size,&SIZE,sizeof(int));
    // printf("**%d\n",cudaMallocManaged((void**)&p1,SIZE));
    hipMallocManaged((void**)&p1,SIZE);
    hipMallocManaged((void**)&vx1, SIZE);
    hipMallocManaged((void**)&vz1, SIZE);
    VX1=pVX1;
    VZ1=pVZ1;
    P1=pP1;
    hipMallocManaged((void**)&z0,SIZE);
    hipDeviceSynchronize();
    //     for(int i = 0; i<ym; i++)
    // {
        
    //     for(int j = 0; j <xm; j++)
    //     {
    //         printf("%f ",p1[j*xar+i*yar]);
    //     }
    //     printf("\n");
        
    // }
    memcpy(z0,Z0,SIZE);
    // memcpy(p1,pP1,size);


    // cudaMemcpy(p1,pP1,size,cudaMemcpyHostToDevice);
    zeroinit<<<ym,xm>>>(p1,xin,yin);
    zeroinit<<<ym,xm>>>(vx1,xin,yin);
    zeroinit<<<ym,xm>>>(vz1,xin,yin);
    hipDeviceSynchronize();
    // memcpy(pP1,p1,SIZE);
    for (int time = 0; time < n; time++)
    {
        // sou<<<1,1>>>(8,4,n,p1,vx1,vz1,z0);

        // cudaDeviceSynchronize();
        p1[200*xar+200*yar]+=sin(0.03*n);
        jisuan1<<<yma-2,xma-2>>>(p1,vx1,vz1,z0,xin,yin);
        hipDeviceSynchronize();
        jisuan2<<<yma-2,xma-2>>>(p1,vx1,vz1,z0,xin,yin);
        hipDeviceSynchronize();
        
    }
    // p1[4*xar+4*yar]=1;
    memcpy(pP1,p1,SIZE);
    // cudaDeviceSynchronize();
    return 0;
}
__global__ void zeroinit(float * date,int xr,int yr) 
{
    int i,j;
    i=threadIdx.x;
    j=blockIdx.x;
    // printf("%d-%d :%f\n",i,j, date[j*yr+i*xr]);
    // printf("%d %d;",xr,yr);
    date[j*yr+i*xr]=0;
    // printf("%s \n","OK!dog" );
    
    
}


__global__ void kkp() 
{
    int i,j;
    i=threadIdx.x;
    j=blockIdx.x;
    printf("%d-%d :\n",i,j);
}


int soinit(float **pP0,int num,int tmax,int *xpl,int *ypl)
{
    for (int i = 0; i < num; i++)
    {
        ;
    }
    return 0;
}

int sdogkel(int n)
{
    int sxp,syp;
    sxp=(int)(xma/2);
    syp=(int)(yma/2);

    for (int i = 0; i < n; i++)
    {
        // sou<<<1,1>>>(sxp,syp,n);
        // jisuan1<<<yma-2,xma-2>>>();
        hipDeviceSynchronize();
        // jisuan2<<<yma-2,xma-2>>>();
        printf("%d \n",i);
    }
    hipDeviceSynchronize();
    hipMemcpy(P1,p1,SIZE,hipMemcpyDeviceToHost);
    printf("%s \n","00" );
    kkp<<<2,2>>>();
    jisuan3<<<yma-2,xma-2>>>();
    hipDeviceSynchronize();
    printf("%s \n","00" );
    printf("%d \n",SIZE);
        for(int i = 0; i<yma; i++)
    {
        
        for(int j = 0; j <xma; j++)
        {
            printf("%f ",P1[j*XIN+i*YIN]);
        }
        printf("%d\n",i);
        
    }
    // printf("****");
    hipMemcpy(P1,p1,SIZE,hipMemcpyDeviceToHost);
    // memcpy(VX1,vx1,SIZE);
    // memcpy(VZ1,vz1,SIZE);
    return 0;
}

__global__ void jisuan1(float * p1j,float *vx1j,float* vz1j,float * z0j,int xr,int yr) 
{
    // printf("%s \n","OKdog!");
    int i,j;
    // printf("%s ","vx1[j*yin+i*xin]");
    i=threadIdx.x+1;
    j=blockIdx.x+1;
    vx1j[j*yr+i*xr]-=(p1j[j*yr+i*xr]-p1j[j*yr+(i-1)*xr])/z0j[j*yr+i*xr]/m;
    vz1j[j*yr+i*xr]-=(p1j[(j+1)*yr+i*xr]-p1j[j*yr+(i)*xr])/z0j[j*yr+i*xr]/m;
    // printf("%s ","vx1[j*yin+i*xin]");
}

__global__ void jisuan2(float * p1j,float *vx1j,float* vz1j,float * z0j,int xr,int yr) 
{
    int i,j;
    i=threadIdx.x+1;
    j=blockIdx.x+1;
    p1j[j*yr+i*xr]-=(vx1j[j*yr+(i+1)*xr]-vx1j[j*yr+i*xr]+vz1j[j*yr+i*xr]-vz1j[(j-1)*yr+i*xr])*z0j[j*yr+i*xr]/m;
    // printf("%f ",p1j[j*yr+i*xr]);
    // if (j==1)
    // {
    //     printf("*%d\n",yr);
    // }
    
    

}

__global__ void sou(int x,int y,int n,float * p1j,float *vx1j,float* vz1j,float * z0j) 
{
    p1j[y*yin+x*xin]+=(float)sin(float(n)/30);
}

__global__ void jisuan3() 
{
    printf("%s \n","OKdog!");
    int i,j;
    i=threadIdx.x+1;
    j=blockIdx.x+1;
    printf("%f ",vx1[j*yin+i*xin]);
}


















__global__ void calkel(float * date,int xr,int yr) 
{
    int i,j;
    i=threadIdx.x;
    j=blockIdx.x;
    // printf("%d-%d :%f\n",i,j, date[j*yr+i*xr]);
    date[j*yr+i*xr]=(float)(i+j)*(i-j);
    
}
int cucaldog(float * date,int xar,int yar,int xm,int ym)
{
    int size;
    float * num=NULL;
    xar=xar/sizeof(float);
    yar=yar/sizeof(float);
    size=(ym*yar)*sizeof(float);
    // printf("%f \n",date[(xm-1)*yar+(ym-1)*xar-1]);
    // for(int i = 0; i < ym; i++)
    // {
    //     for (int j = 0; j < xm; j++)
    //     {
    //         printf("%f@%d ",date[j*xar+i*yar],j*xar+i*yar);
    //         // p[j*xar+i*yar]=(float)sin(i+j);
    //     }
    //     printf("\n");
    // }
    hipMallocManaged((void**)&num, size);
    // cudaMemcpy(num, date, size, cudaMemcpyHostToDevice);
    // printf("***%d \n",xar*yar);
    memcpy(num, date,size);
    calkel<<<ym,xm>>>(num,xar,yar);
    hipDeviceSynchronize();
    memcpy(date,num,size);
    hipFree(num);
    return 0;
}
