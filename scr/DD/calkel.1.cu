
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

extern "C"
{
    int cal(float *P1,float *VX,float *VZ,float *Z0,int xar,int yar,int xm,int ym,int n,int n2,float m,int flag);
}


__global__ void cul1(float * p1j,float * vx1j,float * vz1j,float *z0j,int xr, int yr,float m)
{
    // 计算kel1，用于进行梯度运算。
    // printf("%f \n",m);
    int i,j;
    i=threadIdx.x+1;
    j=blockIdx.x+1;
    vx1j[j*yr+i*xr]-=(p1j[j*yr+i*xr]-p1j[j*yr+(i-1)*xr])/z0j[j*yr+i*xr]/m;
    vz1j[j*yr+i*xr]-=(p1j[(j+1)*yr+i*xr]-p1j[j*yr+(i)*xr])/z0j[j*yr+i*xr]/m;
}


__global__ void cul2(float * p1j,float * vx1j,float * vz1j,float *z0j,int xr, int yr,float m) 
{
    // 计算kel2，用于计算压强。
    int i,j;
    i=threadIdx.x+1;
    j=blockIdx.x+1;
    p1j[j*yr+i*xr]-=(vx1j[j*yr+(i+1)*xr]-vx1j[j*yr+i*xr]+vz1j[j*yr+i*xr]-vz1j[(j-1)*yr+i*xr])*z0j[j*yr+i*xr]/m;
}

int cal(float *P1,float *VX,float *VZ,float *Z0,int xar,int yar,int xm,int ym,int n,int n2,float m,int flag)
{
    // printf("%f %f %f\n",m,P1[400],Z0[400]);
    int size;
    float * p1=NULL;
    float * vx=NULL;
    float * vz=NULL;
    float * z0=NULL;
    
    xar=xar/sizeof(float);
    yar=yar/sizeof(float);
    size=(ym*yar)*sizeof(float);
    // printf("%d %d \n",xar,yar);
    // printf("%d %d %d %d\n",xar,yar,xm,ym);
    // printf("%d \n",size);
    // cudaMallocManaged((void**)&p1, size);
    
    hipMallocManaged((void**)&p1, size);
    hipMallocManaged((void**)&vx, size);
    hipMallocManaged((void**)&vz, size);
    hipMallocManaged((void**)&z0, size);
    hipDeviceSynchronize();
    // printf("%hd \n",p1);
    memcpy(p1,P1,size);
    
    memcpy(p1,P1,size);
    memcpy(vx,VX,size);
    memcpy(vz,VZ,size);
    memcpy(z0,Z0,size);
    // printf("%s \n", "OK");
    hipDeviceSynchronize();
    for (int i = n; i < n2; i++)
    {
        p1[200*xar+200*yar]=sin(0.008*i);
        cul1<<<ym-2,xm-2>>>(p1,vx,vz,z0,xar,yar,m);
        hipDeviceSynchronize();
        cul2<<<ym-2,xm-2>>>(p1,vx,vz,z0,xar,yar,m);
        hipDeviceSynchronize();
    }
    hipDeviceSynchronize();
    // for(int i = 0; i<ym; i++)
    // {
        
    //     for(int j = 0; j <xm; j++)
    //     {
    //         printf("%f ",Z0[j*xar+i*yar]);
    //     }
    //     printf("\n");
        
    // }
    memcpy(P1,p1,size);
    memcpy(VX,vx,size);
    memcpy(VZ,vz,size);

    hipFree(p1);
    hipFree(vx);
    hipFree(vz);
    hipFree(z0);
    
    return 0;
}

