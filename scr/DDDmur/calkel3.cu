
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

extern "C"
{
    int cal(float *P1,float *VX,float *VY,float *VZ,float *Z0,int xar,int yar,int zar,int xm,int ym,int zm,int n,int n2,float m,int flag);
    void usercode(float * p1j,float * vx1j,float * vy1j,float * vz1j,float *z0j,int xr, int yr,int zr,float m,int i);
}

void usercode(float * p1j,float * vx1j,float * vy1j,float * vz1j,float *z0j,int xr, int yr,int zr,float m,int i);

__global__ void cul1(float * p1j,float * vx1j,float * vy1j,float * vz1j,float *z0j,int xr, int yr,int zr,float m)
{
    // 计算kel1，用于进行梯度运算。
    // printf("%f \n",m);
    int i,j,k;
    i=threadIdx.x;
    j=blockIdx.x;
    k=blockIdx.y;

    if ((i>=1)&&(i<=blockDim.x-2)&&(j>=1)&&(j<=gridDim.x-2)&&(k>=1)&&(k<=gridDim.y-2))
    {
        if (z0j[j*yr+i*xr+k*zr]==0)
        {
            vx1j[j*yr+i*xr+k*zr]=0;
            vy1j[j*yr+i*xr+k*zr]=0;
            vz1j[j*yr+i*xr+k*zr]=0;
        }
        else
        {
            vx1j[j*yr+i*xr+k*zr]-=(p1j[j*yr+i*xr+k*zr]-p1j[j*yr+(i-1)*xr+k*zr])/z0j[j*yr+i*xr+k*zr]/m;
            vy1j[j*yr+i*xr+k*zr]-=(p1j[(j+1)*yr+i*xr+k*zr]-p1j[j*yr+(i)*xr+k*zr])/z0j[j*yr+i*xr+k*zr]/m;
            vz1j[j*yr+i*xr+k*zr]-=(p1j[(j)*yr+i*xr+(k+1)*zr]-p1j[j*yr+(i)*xr+k*zr])/z0j[j*yr+i*xr+k*zr]/m;
        }
    }
    else if (i==0)
    {
        p1j[(j)*yr+i*xr+(k)*zr]=p1j[(j)*yr+(i+2)*xr+(k)*zr];
    }

    else if (i==blockDim.x-1)
    {
        p1j[(j)*yr+i*xr+(k)*zr]=p1j[(j)*yr+(i-2)*xr+(k)*zr];
    }

    else if (j==0)
    {
        p1j[(j)*yr+i*xr+(k)*zr]=p1j[(j+2)*yr+(i)*xr+(k)*zr];
    }

    else if (j==gridDim.x-1)
    {
        p1j[(j)*yr+i*xr+(k)*zr]=p1j[(j-2)*yr+(i)*xr+(k)*zr];
    }

    else if (k==0)
    {
        p1j[(j)*yr+i*xr+(k)*zr]=p1j[(j)*yr+(i)*xr+(k+2)*zr];
    }

    else if (k==gridDim.y-1)
    {
        p1j[(j)*yr+i*xr+(k)*zr]=p1j[(j)*yr+(i)*xr+(k-2)*zr];
    }

    

}


__global__ void cul2(float * p1j,float * vx1j,float * vy1j,float * vz1j,float *z0j,int xr, int yr,int zr,float m) 
{
    // 计算kel2，用于计算压强。
    int i,j,k;
    i=threadIdx.x;
    j=blockIdx.x;
    k=blockIdx.y;
    if((i>=1)&&(i<=blockDim.x-2)&&(j>=1)&&(j<=gridDim.x-2)&&(k>=1)&&(k<=gridDim.y-2))
    {
        if((i==1)&&(j>=2)&&(k>=2))
        {
            float dog;
            dog=p1j[j*yr+(i-1)*xr+k*zr]-(vx1j[j*yr+(i+2)*xr+k*zr]-vx1j[j*yr+(i+1)*xr+k*zr]+vy1j[j*yr+(i+1)*xr+k*zr]-vy1j[(j-1)*yr+(i+1)*xr+k*zr]+vz1j[j*yr+(i+1)*xr+k*zr]-vz1j[(j)*yr+(i+1)*xr+(k-1)*zr])*z0j[j*yr+(i+1)*xr+k*zr]/m;
            p1j[j*yr+i*xr+k*zr]=p1j[j*yr+(i-1)*xr+k*zr]+(1.0-m)/(1.0+m)*(dog-p1j[j*yr+i*xr+k*zr]);
        }
        else if ((i==blockDim.x-2)&&(j>=2)&&(k>=2))
        {
            float dog;
            dog=p1j[j*yr+(i+1)*xr+k*zr]-(vx1j[j*yr+(i)*xr+k*zr]-vx1j[j*yr+(i-1)*xr+k*zr]+vy1j[j*yr+(i-1)*xr+k*zr]-vy1j[(j-1)*yr+(i-1)*xr+k*zr]+vz1j[j*yr+(i-1)*xr+k*zr]-vz1j[(j)*yr+(i-1)*xr+(k-1)*zr])*z0j[j*yr+(i-1)*xr+k*zr]/m;
            p1j[j*yr+i*xr+k*zr]=p1j[j*yr+(i+1)*xr+k*zr]+(1.0-m)/(1.0+m)*(dog-p1j[j*yr+i*xr+k*zr]);
        }
        else if ((j==1)&&(i<=blockDim.x-3)&&(k>=2))
        {
            float dog;
            dog=p1j[(j-1)*yr+i*xr+k*zr]-(vx1j[(j+1)*yr+(i+1)*xr+k*zr]-vx1j[(j+1)*yr+i*xr+k*zr]+vy1j[(j+1)*yr+i*xr+k*zr]-vy1j[(j)*yr+i*xr+k*zr]+vz1j[(j+1)*yr+i*xr+k*zr]-vz1j[(j+1)*yr+i*xr+(k-1)*zr])*z0j[(j+1)*yr+i*xr+k*zr]/m;
            p1j[j*yr+i*xr+k*zr]=p1j[(j-1)*yr+i*xr+k*zr]+(1.0-m)/(1.0+m)*(dog-p1j[j*yr+i*xr+k*zr]);
            // if((i<=2)||(i>=blockDim.x-3)||(k<=2)||(k>=gridDim.y-3))
            // {
            //     p1j[j*yr+i*xr+k*zr]=0;
            //     // printf("%s \n","ok");
            // }
            // else
            // {
            //     p1j[j*yr+i*xr+k*zr]=0;//p1j[(j-1)*yr+i*xr+k*zr]+(1.0-m)/(1.0+m)*(dog-p1j[j*yr+i*xr+k*zr]);
            // }
        }
        else if ((j==gridDim.x-2)&&(i<=blockDim.x-3)&&(k>=2))
        {
            float dog;
            dog=p1j[(j+1)*yr+i*xr+k*zr]-(vx1j[(j-1)*yr+(i+1)*xr+k*zr]-vx1j[(j-1)*yr+i*xr+k*zr]+vy1j[(j-1)*yr+i*xr+k*zr]-vy1j[(j-2)*yr+i*xr+k*zr]+vz1j[(j-1)*yr+i*xr+k*zr]-vz1j[(j-1)*yr+i*xr+(k-1)*zr])*z0j[(j-1)*yr+i*xr+k*zr]/m;
            p1j[j*yr+i*xr+k*zr]=p1j[(j+1)*yr+i*xr+k*zr]+(1.0-m)/(1.0+m)*(dog-p1j[j*yr+i*xr+k*zr]);
        }
        else if ((k==1)&&(i<=blockDim.x-3)&&(j>=2))
        {
            float dog;
            dog=p1j[j*yr+i*xr+(k-1)*zr]-(vx1j[j*yr+(i+1)*xr+(k+1)*zr]-vx1j[j*yr+i*xr+(k+1)*zr]+vy1j[j*yr+i*xr+(k+1)*zr]-vy1j[(j-1)*yr+i*xr+(k+1)*zr]+vz1j[j*yr+i*xr+(k+1)*zr]-vz1j[(j)*yr+i*xr+(k)*zr])*z0j[j*yr+i*xr+(k+1)*zr]/m;
            p1j[j*yr+i*xr+k*zr]=p1j[(j)*yr+i*xr+(k-1)*zr]+(1.0-m)/(1.0+m)*(dog-p1j[j*yr+i*xr+k*zr]);
            // 坏事
        }
        else if((k==gridDim.y-2)&&(i<=blockDim.x-3)&&(j>=2))
        {
            float dog;
            dog=p1j[j*yr+i*xr+(k+1)*zr]-(vx1j[j*yr+(i+1)*xr+(k-1)*zr]-vx1j[j*yr+i*xr+(k-1)*zr]+vy1j[j*yr+i*xr+(k-1)*zr]-vy1j[(j-1)*yr+i*xr+(k-1)*zr]+vz1j[j*yr+i*xr+(k-1)*zr]-vz1j[(j)*yr+i*xr+(k-2)*zr])*z0j[j*yr+i*xr+(k-1)*zr]/m;
            p1j[j*yr+i*xr+k*zr]=p1j[(j)*yr+i*xr+(k+1)*zr]+(1.0-m)/(1.0+m)*(dog-p1j[j*yr+i*xr+k*zr]);
            // 坏事
        }
        else if ((i>=2)&&(i<=blockDim.x-3)&&(j>=2)&&(j<=gridDim.x-3)&&(k>=2)&&(k<=gridDim.y-3))
        {
        p1j[j*yr+i*xr+k*zr]-=(vx1j[j*yr+(i+1)*xr+k*zr]-vx1j[j*yr+i*xr+k*zr]+vy1j[j*yr+i*xr+k*zr]-vy1j[(j-1)*yr+i*xr+k*zr]+vz1j[j*yr+i*xr+k*zr]-vz1j[(j)*yr+i*xr+(k-1)*zr])*z0j[j*yr+i*xr+k*zr]/m;
        // printf("%s \n","OK");
        }
    }
}

int cal(float *P1,float *VX,float *VY,float *VZ,float *Z0,int xar,int yar,int zar,int xm,int ym,int zm,int n,int n2,float m,int flag)
{
    // printf("%f %f %f\n",m,P1[400],Z0[400]);
    int size;
    float * p1=NULL;
    float * vx=NULL;
    float * vy=NULL;
    float * vz=NULL;
    float * z0=NULL;
    
    xar=xar/sizeof(float);
    yar=yar/sizeof(float);
    zar=zar/sizeof(float);
    size=(zm*zar)*sizeof(float);
    // printf("%d %d \n",xar,yar);
    // printf("%d %d %d %d\n",xar,yar,xm,ym);
    // printf("%d \n",size);
    // cudaMallocManaged((void**)&p1, size);
    
    hipMallocManaged((void**)&p1, size);
    hipMallocManaged((void**)&vx, size);
    hipMallocManaged((void**)&vy, size);
    hipMallocManaged((void**)&vz, size);
    hipMallocManaged((void**)&z0, size);
    hipDeviceSynchronize();
    // printf("%hd \n",p1);
    memcpy(p1,P1,size);
    
    memcpy(p1,P1,size);
    memcpy(vx,VX,size);
    memcpy(vz,VZ,size);
    memcpy(vy,VY,size);
    memcpy(z0,Z0,size);
    // printf("%s \n", "OK");
    hipDeviceSynchronize();
    dim3 dog(ym,zm);
    for (int i = n; i < n2; i++)
    {
        usercode(p1,vx,vy,vz,z0,xar,yar,zar,m,i);
        // p1[64*xar+64*yar+64*zar]=sin(0.1*i);
        
        cul1<<<dog,xm>>>(p1,vx,vy,vz,z0,xar,yar,zar,m);
        hipDeviceSynchronize();
        cul2<<<dog,xm>>>(p1,vx,vy,vz,z0,xar,yar,zar,m);
        hipDeviceSynchronize();
    }
    hipDeviceSynchronize();
    // for(int i = 0; i<ym; i++)
    // {
        
    //     for(int j = 0; j <xm; j++)
    //     {
    //         printf("%f ",Z0[j*xar+i*yar]);
    //     }
    //     printf("\n");
        
    // }
    memcpy(P1,p1,size);
    memcpy(VX,vx,size);
    memcpy(VZ,vz,size);
    memcpy(VY,vy,size);
    hipFree(p1);
    hipFree(vx);
    hipFree(vz);
    hipFree(vy);
    hipFree(z0);
    hipDeviceReset();
    return 0;
}

// ** //