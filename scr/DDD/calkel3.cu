
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

extern "C"
{
    int cal(float *P1,float *VX,float *VY,float *VZ,float *Z0,int xar,int yar,int zar,int xm,int ym,int zm,int n,int n2,float m,int flag);
    void usercode(float * p1j,float * vx1j,float * vy1j,float * vz1j,float *z0j,int xr, int yr,int zr,float m,int i);
}

void usercode(float * p1j,float * vx1j,float * vy1j,float * vz1j,float *z0j,int xr, int yr,int zr,float m,int i);

__global__ void cul1(float * p1j,float * vx1j,float * vy1j,float * vz1j,float *z0j,int xr, int yr,int zr,float m)
{
    // 计算kel1，用于进行梯度运算。
    // printf("%f \n",m);
    int i,j,k;
    i=threadIdx.x+1;
    j=blockIdx.x+1;
    k=blockIdx.y+1;
    vx1j[j*yr+i*xr+k*zr]-=(p1j[j*yr+i*xr+k*zr]-p1j[j*yr+(i-1)*xr+k*zr])/z0j[j*yr+i*xr+k*zr]/m;
    vy1j[j*yr+i*xr+k*zr]-=(p1j[(j+1)*yr+i*xr+k*zr]-p1j[j*yr+(i)*xr+k*zr])/z0j[j*yr+i*xr+k*zr]/m;
    vz1j[j*yr+i*xr+k*zr]-=(p1j[(j)*yr+i*xr+(k+1)*zr]-p1j[j*yr+(i)*xr+k*zr])/z0j[j*yr+i*xr+k*zr]/m;

}


__global__ void cul2(float * p1j,float * vx1j,float * vy1j,float * vz1j,float *z0j,int xr, int yr,int zr,float m) 
{
    // 计算kel2，用于计算压强。
    int i,j,k;
    i=threadIdx.x+1;
    j=blockIdx.x+1;
    k=blockIdx.y+1;
    p1j[j*yr+i*xr+k*zr]-=(vx1j[j*yr+(i+1)*xr+k*zr]-vx1j[j*yr+i*xr+k*zr]+vy1j[j*yr+i*xr+k*zr]-vy1j[(j-1)*yr+i*xr+k*zr]+vz1j[j*yr+i*xr+k*zr]-vz1j[(j)*yr+i*xr+(k-1)*zr])*z0j[j*yr+i*xr+k*zr]/m;
}

int cal(float *P1,float *VX,float *VY,float *VZ,float *Z0,int xar,int yar,int zar,int xm,int ym,int zm,int n,int n2,float m,int flag)
{
    // printf("%f %f %f\n",m,P1[400],Z0[400]);
    int size;
    float * p1=NULL;
    float * vx=NULL;
    float * vy=NULL;
    float * vz=NULL;
    float * z0=NULL;
    
    xar=xar/sizeof(float);
    yar=yar/sizeof(float);
    zar=zar/sizeof(float);
    size=(zm*zar)*sizeof(float);
    // printf("%d %d \n",xar,yar);
    // printf("%d %d %d %d\n",xar,yar,xm,ym);
    // printf("%d \n",size);
    // cudaMallocManaged((void**)&p1, size);
    
    hipMallocManaged((void**)&p1, size);
    hipMallocManaged((void**)&vx, size);
    hipMallocManaged((void**)&vy, size);
    hipMallocManaged((void**)&vz, size);
    hipMallocManaged((void**)&z0, size);
    hipDeviceSynchronize();
    // printf("%hd \n",p1);
    memcpy(p1,P1,size);
    
    memcpy(p1,P1,size);
    memcpy(vx,VX,size);
    memcpy(vz,VZ,size);
    memcpy(vy,VY,size);
    memcpy(z0,Z0,size);
    // printf("%s \n", "OK");
    hipDeviceSynchronize();
    dim3 dog(ym-2,zm-2);
    for (int i = n; i < n2; i++)
    {
        // p1[128*xar+128*yar+128*zar]=sin(0.008*i);
        usercode(p1,vx,vy,vz,z0,xar,yar,zar,m,i);
        cul1<<<dog,xm-2>>>(p1,vx,vy,vz,z0,xar,yar,zar,m);
        hipDeviceSynchronize();
        cul2<<<dog,xm-2>>>(p1,vx,vy,vz,z0,xar,yar,zar,m);
        hipDeviceSynchronize();
    }
    hipDeviceSynchronize();
    // for(int i = 0; i<ym; i++)
    // {
        
    //     for(int j = 0; j <xm; j++)
    //     {
    //         printf("%f ",Z0[j*xar+i*yar]);
    //     }
    //     printf("\n");
        
    // }
    memcpy(P1,p1,size);
    memcpy(VX,vx,size);
    memcpy(VZ,vz,size);
    memcpy(VY,vy,size);
    hipFree(p1);
    hipFree(vx);
    hipFree(vz);
    hipFree(vy);
    hipFree(z0);
    hipDeviceReset();
    return 0;
}

