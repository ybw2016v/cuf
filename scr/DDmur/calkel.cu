
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

extern "C"
{
    int cal(float *P1,float *VX,float *VZ,float *Z0,float *LX,float *LZ,int xar,int yar,int xm,int ym,int n,int n2,float m,int flag);
    void usercode(float * p1,float * vx,float * vy,float * vz,float *z0,int xar, int yar,int zar,float m,int i);
}
void usercode(float * p1,float * vx1,float * vz1,float *z0j,int xar, int yar,float m,int i);


__global__ void cul1(float * p1j,float * vx1j,float * vz1j,float *z0j,float *lx,float *ly,int xr, int yr,float m)
{
    // 计算kel1，用于进行梯度运算。
    // printf("%f \n",m);
    int i,j;
    i=threadIdx.x;
    j=blockIdx.x;
    if((i>=1)&&(j<gridDim.x-1))
    {
        vx1j[j*yr+i*xr]-=(p1j[j*yr+i*xr]-p1j[j*yr+(i-1)*xr])/z0j[j*yr+i*xr]/m;
        vz1j[j*yr+i*xr]-=(p1j[(j+1)*yr+i*xr]-p1j[j*yr+i*xr])/z0j[j*yr+i*xr]/m;
    }
    if(j==1)
    {
        if (i>0&&j<blockDim.x-1)
        {
            lx[i]=p1j[(j)*yr+i*xr];
        }
        else 
        {
            lx[i]=0;
        }
        
    }
    if(j==gridDim.x-2)
    {
        lx[blockDim.x+i]=p1j[(j)*yr+i*xr];
    }
    if (i==1)
    {

        ly[j]=p1j[(j)*yr+i*xr];
    }
    if (i==blockDim.x-2)
    {
        ly[gridDim.x+j]=p1j[(j)*yr+i*xr];
    }

}


__global__ void cul2(float * p1j,float * vx1j,float * vz1j,float *z0j,float *lx,float *ly,int xr, int yr,float m) 
{
    // 计算kel2，用于计算压强。
    int i,j;
    i=threadIdx.x;
    j=blockIdx.x;
    if (j==0)
    {
        float dog;
        dog=lx[i]-(vx1j[(j+1)*yr+(i+1)*xr]-vx1j[(j+1)*yr+i*xr]+vz1j[(j+1)*yr+i*xr]-vz1j[(j)*yr+(i)*xr])*z0j[j*yr+i*xr]/m;
        p1j[j*yr+i*xr]=lx[i]+(1.0-m)/(1.0+m)*(dog-p1j[j*yr+i*xr]);    
    }

    if(j==gridDim.x-1)
    {
        float dog;
        dog=lx[blockDim.x+i]-(vx1j[(j-1)*yr+(i+1)*xr]-vx1j[(j-1)*yr+i*xr]+vz1j[(j-1)*yr+i*xr]-vz1j[(j-2)*yr+(i)*xr])*z0j[j*yr+i*xr]/m;
        p1j[j*yr+i*xr]=lx[blockDim.x+i]+(1.0-m)/(1.0+m)*(dog-p1j[j*yr+i*xr]); 
    }
    if (i==0)
    {
        float dog;
        dog=ly[j]-(vx1j[j*yr+(i+2)*xr]-vx1j[j*yr+(i+1)*xr]+vz1j[j*yr+(i+1)*xr]-vz1j[(j-1)*yr+(i+1)*xr])*z0j[j*yr+i*xr]/m;
        p1j[j*yr+i*xr]=ly[j]+(1.0-m)/(1.0+m)*(dog-p1j[j*yr+i*xr]);    

    }
    if (i==blockDim.x-1)
    {
        float dog;
        dog=lx[gridDim.x+j]-(vx1j[j*yr+(i)*xr]-vx1j[j*yr+(i-1)*xr]+vz1j[j*yr+(i-1)*xr]-vz1j[(j-1)*yr+(i-1)*xr])*z0j[j*yr+i*xr]/m;
        p1j[j*yr+i*xr]=lx[blockDim.x+j]+(1.0-m)/(1.0+m)*(dog-p1j[j*yr+i*xr]);
    }
    if((i<blockDim.x-1)&&(i>0)&&(j>0)&&(j<gridDim.x-1))
    {
    p1j[j*yr+i*xr]-=(vx1j[j*yr+(i+1)*xr]-vx1j[j*yr+i*xr]+vz1j[j*yr+i*xr]-vz1j[(j-1)*yr+i*xr])*z0j[j*yr+i*xr]/m;
    }

}

int cal(float *P1,float *VX,float *VZ,float *Z0,float *LX,float *LZ,int xar,int yar,int xm,int ym,int n,int n2,float m,int flag)
{
    // printf("%f %f %f\n",m,P1[400],Z0[400]);
    // printf("%s \n","OK");
    int size;
    float * p1=NULL;
    float * vx=NULL;
    float * vz=NULL;
    float * z0=NULL;
    float * lx=NULL;
    float * lz=NULL;

    
    xar=xar/sizeof(float);
    yar=yar/sizeof(float);
    size=(ym*yar)*sizeof(float);
    // printf("%d %d \n",xar,yar);
    // printf("%d %d %d %d\n",xar,yar,xm,ym);
    // printf("%d \n",size);
    // cudaMallocManaged((void**)&p1, size);
    
    hipMallocManaged((void**)&p1, size);
    hipMallocManaged((void**)&vx, size);
    hipMallocManaged((void**)&vz, size);
    hipMallocManaged((void**)&z0, size);
    hipMallocManaged((void**)&lx, 2*xm*sizeof(float));
    hipMallocManaged((void**)&lz, 2*ym*sizeof(float));
    
    hipDeviceSynchronize();
    // printf("%hd \n",p1);
    memcpy(p1,P1,size);
    
    memcpy(p1,P1,size);
    memcpy(vx,VX,size);
    memcpy(vz,VZ,size);
    memcpy(z0,Z0,size);
    memcpy(lx,LX,2*xm*sizeof(float));
    memcpy(lz,LZ,2*ym*sizeof(float));
    // printf("%s \n", "OK");
    hipDeviceSynchronize();
    for (int i = n; i < n2; i++)
    {
        usercode(p1,vx,vz,z0,xar,yar,m,i);
        //p1[200*xar+200*yar]=sin(0.08*i);
        cul1<<<ym,xm>>>(p1,vx,vz,z0,lx,lz,xar,yar,m);
        hipDeviceSynchronize();
        cul2<<<ym,xm>>>(p1,vx,vz,z0,lx,lz,xar,yar,m);
        hipDeviceSynchronize();
    }
    hipDeviceSynchronize();
    // for(int i = 0; i<ym; i++)
    // {
        
    //     for(int j = 0; j <xm; j++)
    //     {
    //         printf("%f ",Z0[j*xar+i*yar]);
    //     }
    //     printf("\n");
        
    // }
    memcpy(P1,p1,size);
    memcpy(VX,vx,size);
    memcpy(VZ,vz,size);
    memcpy(LX,lx,2*xm*sizeof(float));
    memcpy(LZ,lz,2*ym*sizeof(float));
    hipFree(p1);
    hipFree(vx);
    hipFree(vz);
    hipFree(z0);
    hipFree(lx);
    hipFree(lz);
    
    return 0;
}

